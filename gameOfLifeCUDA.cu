#include "hip/hip_runtime.h"

#include "parameter.h"
#include "utils.h"
#include <time.h> 
#include "CycleTimer.h"
#include <unistd.h>
#include <hip/hip_runtime.h>
__global__ void updateCUDAKernel(bool* gridOne, bool* gridTwo){
    uint worldSize = gridWidth * gridHeight;
    for(uint cellID = blockIdx.x * blockDim.x + threadIdx.x;
        cellID < worldSize;
        cellID += blockDim.x * gridDim.x){
        uint x = (cellID % (gridWidth)) + 1; // 1 base
        uint y = ((cellID - (cellID % (gridWidth)))/gridWidth)*arrayWidth + arrayWidth;  // 1 base
        uint xLeft = x - 1;
        uint xRight = x + 1;
        uint yUp = y - arrayWidth;
        uint yDown = y + arrayWidth;

        uint alive =      gridTwo[xLeft + yUp]   + gridTwo[x + yUp]   + gridTwo[xRight + yUp] +
                          gridTwo[xLeft + y]     +                    + gridTwo[xRight + y] +
                          gridTwo[xLeft + yDown] + gridTwo[x + yDown] + gridTwo[xRight + yDown];
        
        gridOne[x + y] = alive == 3 || (alive == 2 && gridTwo[x + y]) ? 1 : 0 ;   
    }
}

double gameOfLifeCUDA(bool* &gridOne, bool* &gridTwo, char mode){
    
    int i = 0;
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);

    initGrid(mode, gridOne);
    

    int size = arrayHeight * arrayWidth;
    bool *d_gridOne, *d_gridTwo;

    int iter = 0;  
    float elapseTime = 0.0;
    size_t threadCount = min(128, size);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    hipMalloc(&d_gridOne, size*sizeof(bool));
    hipMalloc(&d_gridTwo, size*sizeof(bool));
    hipMemcpy(d_gridOne, gridOne, size, hipMemcpyHostToDevice);
    hipMemcpy(d_gridTwo, gridTwo, size, hipMemcpyHostToDevice);
    while (iter++ < maxIteration) 
    {
        std::swap(d_gridOne, d_gridTwo);
        size_t reqBlocksCount = ((gridWidth) * (gridHeight)) / threadCount;
        updateCUDAKernel<<<reqBlocksCount, threadCount>>>(d_gridOne, d_gridTwo);
    }
    hipMemcpy(gridOne, d_gridOne, size, hipMemcpyDeviceToHost);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapseTime, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
   
    hipFree(d_gridOne);
    hipFree(d_gridTwo);
    return elapseTime;
}
